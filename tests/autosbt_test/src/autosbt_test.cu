#include "hip/hip_runtime.h"
#include "autosbt_test.h"

extern "C" {
    __constant__ Params params;
}

extern "C" __global__ void __raygen__autosbt_test()
{
    auto idx  = optixGetLaunchIndex();
    auto dims = optixGetLaunchDimensions();

    float3 rayOrigin, rayDirection;
    params.cam.compute_ray(idx, dims, rayOrigin, rayDirection);

    RgbRay ray;
    ray.trace(params.topObject, rayOrigin, rayDirection);
    params.output[params.width*idx.y + idx.x] = ray;
}

extern "C" __global__ void __miss__autosbt_rgb()
{
    auto data = reinterpret_cast<const RgbMissData*>(optixGetSbtDataPointer());
    RgbRay::set_payload(data->color);
}

extern "C" __global__ void __closesthit__autosbt_rgb()
{
    auto data = reinterpret_cast<const RgbHitData*>(optixGetSbtDataPointer());
    
    float3 hitPosition = optixTransformPointFromObjectToWorldSpace(
        helpers::get_triangle_hit_position());

    // sending a shadow ray towards light starting hitPosition
    float3 rayDirection = normalized(data->light - hitPosition);
    ShadowRay sray;
    sray.trace(params.topObject, hitPosition, rayDirection, 1.0e-4);
    if(sray.tHit < 0.0f) {
        // the shadow ray did not encounter any object. No shadow
        RgbRay::set_payload(data->color);
    }
    else {
        // the shadow ray did encounter an object. Shadow
        uchar3 c = data->color;
        c.x /= 8;
        c.y /= 8;
        c.z /= 8;
        RgbRay::set_payload(c);
    }
}

extern "C" __global__ void __miss__autosbt_shadow()
{
    ShadowRay::set_payload({-1.0f});
}

extern "C" __global__ void __closesthit__autosbt_shadow()
{
    ShadowRay::set_payload({optixGetRayTmax()});
}
