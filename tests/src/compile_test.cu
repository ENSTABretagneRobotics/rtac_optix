#include "hip/hip_runtime.h"
#include <optix.h>
//#include <thrust/device_ptr.h>
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>

#include "compile_test.h"

__device__
void device_copy(const float& v1, float& v2)
{
    v2 = v1;
}

__global__
void global_copy(const float* input, float* output)
{
    device_copy(input[threadIdx.x], output[threadIdx.x]);
}

//void copy(const thrust::device_vector<float>& input,
//          thrust::device_vector<float>& output)
//{
//    global_copy<<<1,input.size()>>>(thrust::raw_pointer_cast(input.data()),
//                                 thrust::raw_pointer_cast(output.data()));
//    hipDeviceSynchronize();
//}

void copy(const std::vector<float>& input, std::vector<float>& output)
{
    //thrust::device_vector<float> in(input);
    //thrust::device_vector<float> out(in.size());
    //
    //copy(in, out);

    //output.resize(out.size());
    //hipMemcpy(output.data(), thrust::raw_pointer_cast(out.data()),
    //           sizeof(float)*out.size(), hipMemcpyDeviceToHost);

    float* inPtr;
    float* outPtr;

    hipMalloc(&inPtr,  input.size()*sizeof(float));
    hipMalloc(&outPtr, output.size()*sizeof(float));

    hipMemcpy(inPtr, input.data(), input.size()*sizeof(float),
               hipMemcpyHostToDevice);

    global_copy<<<1,input.size()>>>(inPtr, outPtr);
    hipDeviceSynchronize();

    hipMemcpy(output.data(), outPtr, output.size()*sizeof(float),
               hipMemcpyDeviceToHost);
}




